
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// a kernel that will turn 1 to 42 (ascii for *)
__global__
void pop_kernel(int n, float *x)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) x[i] = 42.0*x[i];
}

int main() {
  // use the first cuda device you can find.
  hipSetDevice(0);

  // let's pop about a million kernels, allocate memory
  // and set values to 1 (1 means the heat is on)
  int N = 1<<20;
  float *x, *d_x;
  x = (float*)malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float)); 
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
  }

  // copy a million 1's to the GPU
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);

  // set up and start a timer
  float gpu_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // run the kernel on 1M elements
  pop_kernel<<<(N+255)/256, 256>>>(N, d_x);

  // stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpu_time, start, stop);

  // get your popcorn off the GPU
  hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

  printf("Time to generate on GPU:  %3.1f ms \n", gpu_time);

  hipFree(d_x);
  free(x);

  // Now let's try doing the same thing on the CPU

  // allocate fresh memory and set up timers
  x = (float*)malloc(N*sizeof(float));
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
  }
  struct timeval t1, t2;
  gettimeofday(&t1, 0);
  
  // pop that corn on the CPU
  for (int i = 0; i < N; i++) {
    x[i] = 42*x[i];
  }

  gettimeofday(&t2, 0);
  double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

  printf("Time to generate on CPU:  %3.1f ms \n", time);
}
